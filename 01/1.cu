#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define N 1000
#define BLOCK_SIZE 32

__managed__ int input_Matrix[N][N];
__managed__ int output_GPU[N][N];
__managed__ int output_CPU[N][N];
__global__ void gpu_kernel(int input_M[N][N], int output_M[N][N])
{
    int row=blockIdx.y*blockDim.y + threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    if (col < N && row <N){
        if(col%2==0 and row%2==0){
            output_M[col][row] = input_M[col][row]*input_M[col][row];
        }
        else{
            output_M[col][row] = input_M[col][row]-1;
        }
    }
}
void cpu_kernel(int intput_M[N][N], int output_CPU[N][N])
{
    for(int i=0; i<N; i++)
    {
        for(int j=0; j<N; j++)
        {
            if(j%2==0 && i%2==0)
            {
                output_CPU[i][j] = intput_M[i][j]*intput_M[i][j];
            }
            else
            {
                output_CPU[i][j] = intput_M[i][j]-1;
            }
        }
    }
}

int main(int argc, char const *argv[])
{
    
    hipEvent_t start,stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) 
        {
            
            input_Matrix[i][j] = rand()%3001;
            //printf("%d ",input_Matrix[i][j]);
        }
        //printf("\n");
    }
    cpu_kernel(input_Matrix, output_CPU);
    
    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    printf("\n***********GPU RUN**************\n");
    gpu_kernel<<<dimGrid, dimBlock>>>(input_Matrix, output_GPU);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    
    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    for (int i = 0; i < N; ++i)
    { 
        for (int j = 0; j < N; ++j)
        {
            //printf("%d ",output_GPU[i][j]);
            if(fabs(output_GPU[i][j] - output_CPU[i][j])>(1.0e-10))
            {
                ok = 0;
            }
            
        }
        //printf("\n");
    }
    printf("\n***********Check result**************\n");
    for (int i = 0; i < N; ++i)
    { 
        for (int j = 0; j < N; ++j)
        {
            //printf("%d ",output_CPU[i][j]);
            if(fabs(output_GPU[i][j] - output_CPU[i][j])>(1.0e-10))
            {
                ok = 0;
            }
            
        }
        //printf("\n");
    }


    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    
    // free memory
    return 0;
}