#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define BLOCK_SIZE 256
#define N 1000000
#define GRID_SIZE  ((N + BLOCK_SIZE - 1) / BLOCK_SIZE) 
#define lich wuyaowang
#define wuyaowang huanhuan
#define huanhuan happy


__managed__ int sourse_array[N]; 
__managed__ int _1pass_results[2*GRID_SIZE];
__managed__ int final_results[2]; 

__global__ void top_2(int* input, int length, int* output)
{
    __shared__ int lich[BLOCK_SIZE*2];
    int top1 = INT_MIN;
    int top2 = INT_MIN;
    
    for(int idx = threadIdx.x + blockDim.x * blockIdx.x; idx<length; idx+= gridDim.x*blockDim.x)
    {
        if(input[idx]>top2)
        {
            top2 = min(input[idx],top1);
            top1 = max(input[idx],top1);
        }
    }
    lich[2*threadIdx.x]=top1;
    lich[2*threadIdx.x+1]=top2;
    __syncthreads();
    
    int top1_final=0;
    int top2_final=0;
    for (int i = BLOCK_SIZE / 2; i >= 1; i /= 2)
    {
        if(threadIdx.x < i)
        {
            top1_final = max(lich[2*threadIdx.x],lich[2*(threadIdx.x+i)]);
            top2_final = min(max(lich[2*threadIdx.x],lich[2*(threadIdx.x+i)+1]),max(lich[2*threadIdx.x+1],lich[2*(threadIdx.x+i)]));
        }
        __syncthreads();
        
        if(threadIdx.x < i)
        {
            lich[2*threadIdx.x] = top1_final;
            lich[2*threadIdx.x+1]=top2_final;
        }
        __syncthreads();
    }
    if(blockIdx.x*blockDim.x < length)
    {
        if(threadIdx.x == 0)
        {
            output[2*blockIdx.x] = lich[0];
            output[2*blockIdx.x+1] = lich[1];
        }
    }
}

void cpu_result_top2(int* input, int count, int* output)
{
    int top1 = 0;
    int top2 = 0;
    for(int i =0; i<count; i++)
    {
        if(input[i]>top2)
        {
            
            top2 = min(input[i],top1);
            top1 = max(input[i],top1);
        }
    }
    output[0] = top1;
    output[1] = top2;

}

int main(int argc, char const *argv[])
{
    int cpu_result[2] = {0};
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    
    //Fill input data buffer
    for (int i = 0; i < N; ++i)
    {
        sourse_array[i] = rand();
    }
    
    printf("\n***********GPU RUN**************\n");
    CHECK(hipEventRecord(start));
    top_2<<<GRID_SIZE, BLOCK_SIZE>>>(sourse_array, N, _1pass_results);
    CHECK(hipGetLastError());
    top_2<<<1, BLOCK_SIZE>>>(_1pass_results, 2*GRID_SIZE, final_results);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time = %g ms.\n", elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    
    cpu_result_top2(sourse_array, N, cpu_result);

    int ok = 1;
    for (int i = 0; i < 2; ++i)
    {
        printf("cpu top%d: %d; gpu top%d: %d \n", i+1, cpu_result[i], i+1, final_results[i]);
        if(fabs(cpu_result[i] - final_results[i])>(1.0e-10))
        {
                
            ok = 0;
        }
    }

    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    return 0;
}