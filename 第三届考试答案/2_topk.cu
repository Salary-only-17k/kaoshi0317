#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>   
#include "error.cuh"

#define BLOCK_SIZE 256
#define N 1000000
#define GRID_SIZE  ((N + BLOCK_SIZE - 1) / BLOCK_SIZE) 
#define topk 10


__managed__ int source_array[N];
__managed__ int _1pass_results[topk * GRID_SIZE];
__managed__ int final_results[topk];

__device__ __host__ void insert_value(int* array, int k, int data)
{
    for (int i = 0; i < k; i++)
    {
        if (array[i] == data)
        {
            return;
        }
    }
    if (data < array[k - 1])
        return;
    for (int i = k - 2; i >= 0; i--)
    {
        if (data > array[i])
            array[i + 1] = array[i];
        else {
            array[i + 1] = data;
            return;
        }
    }
    array[0] = data;
}

__global__ void top_k(int* input, int length, int* output, int k)
{
    __shared__ int lich[BLOCK_SIZE * topk];
    int top_array[topk];

    for (int i = 0; i < topk; i++)
    {
        top_array[i] = INT_MIN;
    }

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < length; idx += gridDim.x * blockDim.x)
    {
        insert_value(top_array, k, input[idx]);
    }
//#pragma unroll 5
    for (int j = 0; j < topk; j++)
    {
        lich[topk * threadIdx.x+j] = top_array[j];
    }
    __syncthreads();

    for (int i = BLOCK_SIZE / 2; i >= 1; i /= 2)
    {
        if (threadIdx.x < i)
        {
            for (int m = 0; m < topk; m++)
            {
                insert_value(top_array, topk, lich[topk * (threadIdx.x + i) + m]);
            }
        }
        __syncthreads();

        if (threadIdx.x < i)
        {
//#pragma unroll 5
            for (int m = 0; m < topk; m++)
            {
                lich[topk* threadIdx.x + m] = top_array[m];
            }
        }
        __syncthreads();
    }
    if (blockIdx.x * blockDim.x < length)
    {
        if (threadIdx.x == 0)
        {
//#pragma unroll 5
            for (int m = 0; m < topk; m++)
            {
                output[topk * blockIdx.x + m] = lich[m];
            }
        }
    }
}

void cpu_result_topk(int* input, int count, int* output)
{
    /*for (int i = 0; i < topk; i++)
    {
        output[i] = INT_MIN;
    }*/
    for (int i = 0; i < count; i++)
    {
        insert_value(output, topk, input[i]);

    }
}

void _init(int* ptr, int count)
{
    srand((unsigned)time(NULL));
    for (int i = 0; i < count; i++) ptr[i] = rand();
}

int main(int argc, char const* argv[])
{
    int cpu_result[topk] = { 0 };
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    //Fill input data buffer
    _init(source_array, N);


    printf("\n***********GPU RUN**************\n");
    CHECK(hipEventRecord(start));
    top_k << <GRID_SIZE, BLOCK_SIZE >> > (source_array, N, _1pass_results, topk);
    CHECK(hipGetLastError());
    top_k << <1, BLOCK_SIZE >> > (_1pass_results, topk * GRID_SIZE, final_results, topk);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Time = %g ms.\n", elapsed_time);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    cpu_result_topk(source_array, N, cpu_result);

    int ok = 1;
    for (int i = 0; i < topk; ++i)
    {
        printf("cpu top%d: %d; gpu top%d: %d \n", i + 1, cpu_result[i], i + 1, final_results[i]);
        if (fabs(cpu_result[i] - final_results[i]) > (1.0e-10))
        {

            ok = 0;
        }
    }

    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    return 0;
}